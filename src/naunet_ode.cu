#include "hip/hip_runtime.h"
#include <math.h>
/* */
#include <nvector/nvector_cuda.h>
#include <sunmatrix/sunmatrix_cusparse.h>
/* */
/*  */
#include "naunet_ode.h"
/*  */
#include "naunet_constants.h"
#include "naunet_macros.h"
#include "naunet_physics.h"

#define IJth(A, i, j) SM_ELEMENT_D(A, i, j)

// clang-format off
__device__ int EvalRates(realtype *k, realtype *y, NaunetData *u_data) {

    realtype nH = u_data->nH;
    realtype Tgas = u_data->Tgas;
    
    
    // clang-format on

    // Some variable definitions from krome
    realtype Te      = Tgas * 8.617343e-5;            // Tgas in eV (eV)
    realtype lnTe    = log(Te);                       // ln of Te (#)
    realtype T32     = Tgas * 0.0033333333333333335;  // Tgas/(300 K) (#)
    realtype invT    = 1.0 / Tgas;                    // inverse of T (1/K)
    realtype invTe   = 1.0 / Te;                      // inverse of T (1/eV)
    realtype sqrTgas = sqrt(Tgas);  // Tgas rootsquare (K**0.5)

    // reaaction rate (k) of each reaction
    // clang-format off
    k[0] = exp(-32.71396786e0 + 13.5365560e0*lnTe-5.73932875e0*(pow(lnTe,
        2)) + 1.56315498e0*(pow(lnTe, 3)) - 0.28770560e0*(pow(lnTe, 4)) +
        3.48255977e-2*(pow(lnTe, 5)) - 2.63197617e-3*(pow(lnTe, 6)) +
        1.11954395e-4*(pow(lnTe, 7)) - 2.03914985e-6*(pow(lnTe, 8)));
        
    if (Tgas>1.0 && Tgas<5500.0) { k[1] = 3.92e-13*pow(invTe, 0.6353e0);  }
        
    k[2] = exp(-28.61303380689232e0 -
        0.7241125657826851e0*lnTe-0.02026044731984691e0*pow(lnTe, 2) -
        0.002380861877349834e0*pow(lnTe, 3) - 0.0003212605213188796e0*pow(lnTe,
        4) - 0.00001421502914054107e0*pow(lnTe, 5) +
        4.989108920299513e-6*pow(lnTe, 6) + 5.755614137575758e-7*pow(lnTe, 7) -
        1.856767039775261e-8*pow(lnTe, 8) - 3.071135243196595e-9*pow(lnTe, 9));
        
    k[3] = exp(-44.09864886e0 + 23.91596563e0*lnTe-10.7532302e0*(pow(lnTe,
        2)) + 3.05803875e0*(pow(lnTe, 3)) - 0.56851189e0*(pow(lnTe, 4)) +
        6.79539123e-2*(pow(lnTe, 5)) - 5.00905610e-3*(pow(lnTe, 6)) +
        2.06723616e-4*(pow(lnTe, 7)) - 3.64916141e-6*(pow(lnTe, 8)));
        
    if (Tgas>1.0 && Tgas<9280.0) { k[4] = 3.92e-13*pow(invTe, 0.6353e0);  }
        
    k[5] = 1.54e-9*(1.e0 +
        0.3e0/exp(8.099328789667e0*invTe))/(exp(40.49664394833662e0*invTe)*pow(Te,
        1.5e0)) + 3.92e-13/pow(Te, 0.6353e0);
        
    k[6] = exp(-68.71040990212001e0 +
        43.93347632635e0*lnTe-18.48066993568e0*pow(lnTe, 2) +
        4.701626486759002e0*pow(lnTe, 3) - 0.7692466334492e0*pow(lnTe, 4) +
        0.08113042097303e0*pow(lnTe, 5) - 0.005324020628287001e0*pow(lnTe, 6) +
        0.0001975705312221e0*pow(lnTe, 7) - 3.165581065665e-6*pow(lnTe, 8));
        
    k[7] = 3.36e-10/sqrTgas/pow((Tgas/1.e3), 0.2e0)/(1 + pow((Tgas/1.e6),
        0.7e0));
        
    k[8] = 6.77e-15*pow(Te, 0.8779e0);
        
    if (Tgas>1.0 && Tgas<1160.0) { k[9] = 1.43e-9;  }
        
    k[10] = exp(-20.06913897587003e0 +
        0.2289800603272916e0*lnTe+0.03599837721023835e0*pow(lnTe, 2) -
        0.004555120027032095e0*pow(lnTe, 3) - 0.0003105115447124016e0*pow(lnTe,
        4) + 0.0001073294010367247e0*pow(lnTe, 5) -
        8.36671960467864e-6*pow(lnTe, 6) + 2.238306228891639e-7*pow(lnTe, 7));
        
    if (Tgas>1.0 && Tgas<6700.0) { k[11] = 1.85e-23*pow(Tgas, 1.8e0);  }
        
    k[12] = 5.81e-16*pow((Tgas/5.62e4), (-0.6657e0*log10(Tgas/5.62e4)));
        
    k[13] = 6.0e-10;
        
    k[14] = exp(-24.24914687731536e0 +
        3.400824447095291e0*lnTe-3.898003964650152e0*pow(lnTe, 2) +
        2.045587822403071e0*pow(lnTe, 3) - 0.5416182856220388e0*pow(lnTe, 4) +
        0.0841077503763412e0*pow(lnTe, 5) - 0.007879026154483455e0*pow(lnTe, 6)
        + 0.0004138398421504563e0*pow(lnTe, 7) - 9.36345888928611e-6*pow(lnTe,
        8));
        
    k[15] = 5.6e-11*exp(-102124.e0*invT)*pow(Tgas, 0.5e0);
        
    k[16] = 1.0670825e-10*pow(Te, 2.012e0)*exp(-4.463e0*invTe)/pow((1.e0 +
        0.2472e0*Te), 3.512e0);
        
    k[17] = exp(-18.01849334273e0 +
        2.360852208681e0*lnTe-0.2827443061704e0*pow(lnTe, 2) +
        0.01623316639567e0*pow(lnTe, 3) - 0.03365012031362999e0*pow(lnTe, 4) +
        0.01178329782711e0*pow(lnTe, 5) - 0.001656194699504e0*pow(lnTe, 6) +
        0.0001068275202678e0*pow(lnTe, 7) - 2.631285809207e-6*pow(lnTe, 8));
        
    if (Tgas>1.0 && Tgas<1160.0) { k[18] = 2.56e-9*pow(Te, 1.78186e0);  }
        
    k[19] = exp(-20.37260896533324e0 +
        1.139449335841631e0*lnTe-0.1421013521554148e0*pow(lnTe, 2) +
        0.00846445538663e0*pow(lnTe, 3) - 0.0014327641212992e0*pow(lnTe, 4) +
        0.0002012250284791e0*pow(lnTe, 5) + 0.0000866396324309e0*pow(lnTe, 6) -
        0.00002585009680264e0*pow(lnTe, 7) + 2.4555011970392e-6*pow(lnTe, 8) -
        8.06838246118e-8*pow(lnTe, 9));
        
    k[20] = 6.5e-9/sqrt(Te);
        
    k[21] = 1.e-8*pow(Tgas, (-0.4e0));
        
    if (Tgas>1.0 && Tgas<617.0) { k[22] = 1.e-8;  }
        
    k[23] = 1.32e-6*pow(Tgas, (-0.76e0));
        
    k[24] = 5.e-7*sqrt(1.e2*invT);
        
    if (Tgas>1.0 && Tgas<300.0) { k[25] = 1.3e-32*pow((T32), (-0.38e0));  }
        
    k[26] = 1.3e-32*pow((T32), (-1.00e0));
        
    if (Tgas>1.0 && Tgas<300.0) { k[27] = 1.3e-32*pow((T32),
        (-0.38e0))/8.e0;  }
        
    k[28] = 1.3e-32*pow((T32), (-1.00e0))/8.e0;
        
    k[29] = 2.00e-10*pow(Tgas, (0.402e0))*exp(-37.1e0*invT) -
        3.31e-17*pow(Tgas, (1.48e0));
        
    k[30] = 2.06e-10*pow(Tgas, (0.396))*exp(-33.e0*invT) + 2.03e-9*pow(Tgas,
        (-0.332));
        
    k[31] = 1.e-9*(0.417 + 0.846*log10(Tgas) - 0.137*pow((log10(Tgas)), 2));
        
    k[32] = 1.0e-9*exp(-4.57e2*invT);
        
    if (Tgas>1.0 && Tgas<2000.0) { k[33] = pow(10, (-56.4737 +
        5.88886*log10(Tgas) + 7.19692*pow((log10(Tgas)), 2) +
        2.25069*pow((log10(Tgas)), 3) - 2.16903*pow((log10(Tgas)), 4) +
        0.317887*pow((log10(Tgas)), 5)));  }
        
    k[34] = 3.17e-10*exp(-5207.*invT);
        
    k[35] = 5.25e-11*exp(-4430.*invT + 1.739e5*pow((invT), 2));
        
    k[36] = 1.5e-9*pow((T32), (-0.1e0));
        
    k[37] = 3.6e-12*pow((Tgas/300), (-0.75e0));
        
    
        // clang-format on

    return NAUNET_SUCCESS;
}

/* */
int InitJac(SUNMatrix jmatrix) {
    int rowptrs[NEQUATIONS + 1], colvals[NNZ];

    // Zero out the Jacobian
    SUNMatZero(jmatrix);

    // clang-format off
    // number of non-zero elements in each row
    rowptrs[0] = 0;
    rowptrs[1] = 8;
    rowptrs[2] = 15;
    rowptrs[3] = 20;
    rowptrs[4] = 29;
    rowptrs[5] = 38;
    rowptrs[6] = 44;
    rowptrs[7] = 53;
    rowptrs[8] = 59;
    rowptrs[9] = 66;
    rowptrs[10] = 69;
    rowptrs[11] = 73;
    rowptrs[12] = 76;
    rowptrs[13] = 87;
    rowptrs[14] = 93;
    
    // the column index of non-zero elements
    colvals[0] = 0;
    colvals[1] = 1;
    colvals[2] = 3;
    colvals[3] = 4;
    colvals[4] = 5;
    colvals[5] = 6;
    colvals[6] = 8;
    colvals[7] = 12;
    colvals[8] = 0;
    colvals[9] = 1;
    colvals[10] = 3;
    colvals[11] = 4;
    colvals[12] = 6;
    colvals[13] = 8;
    colvals[14] = 12;
    colvals[15] = 3;
    colvals[16] = 4;
    colvals[17] = 10;
    colvals[18] = 11;
    colvals[19] = 12;
    colvals[20] = 0;
    colvals[21] = 1;
    colvals[22] = 3;
    colvals[23] = 4;
    colvals[24] = 5;
    colvals[25] = 6;
    colvals[26] = 7;
    colvals[27] = 8;
    colvals[28] = 12;
    colvals[29] = 0;
    colvals[30] = 1;
    colvals[31] = 3;
    colvals[32] = 4;
    colvals[33] = 5;
    colvals[34] = 6;
    colvals[35] = 7;
    colvals[36] = 8;
    colvals[37] = 12;
    colvals[38] = 0;
    colvals[39] = 3;
    colvals[40] = 4;
    colvals[41] = 5;
    colvals[42] = 7;
    colvals[43] = 12;
    colvals[44] = 0;
    colvals[45] = 1;
    colvals[46] = 3;
    colvals[47] = 4;
    colvals[48] = 5;
    colvals[49] = 6;
    colvals[50] = 7;
    colvals[51] = 8;
    colvals[52] = 12;
    colvals[53] = 3;
    colvals[54] = 4;
    colvals[55] = 5;
    colvals[56] = 6;
    colvals[57] = 7;
    colvals[58] = 12;
    colvals[59] = 0;
    colvals[60] = 1;
    colvals[61] = 3;
    colvals[62] = 4;
    colvals[63] = 5;
    colvals[64] = 6;
    colvals[65] = 8;
    colvals[66] = 9;
    colvals[67] = 10;
    colvals[68] = 12;
    colvals[69] = 9;
    colvals[70] = 10;
    colvals[71] = 11;
    colvals[72] = 12;
    colvals[73] = 10;
    colvals[74] = 11;
    colvals[75] = 12;
    colvals[76] = 0;
    colvals[77] = 1;
    colvals[78] = 3;
    colvals[79] = 4;
    colvals[80] = 5;
    colvals[81] = 6;
    colvals[82] = 7;
    colvals[83] = 9;
    colvals[84] = 10;
    colvals[85] = 11;
    colvals[86] = 12;
    colvals[87] = 3;
    colvals[88] = 4;
    colvals[89] = 9;
    colvals[90] = 10;
    colvals[91] = 11;
    colvals[92] = 12;
    
    // clang-format on

    // copy rowptrs, colvals to the device
    SUNMatrix_cuSparse_CopyToDevice(jmatrix, NULL, rowptrs, colvals);
    hipDeviceSynchronize();

    return NAUNET_SUCCESS;
}

__global__ void FexKernel(realtype *y, realtype *ydot, NaunetData *d_udata,
                          int nsystem) {
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gs   = blockDim.x * gridDim.x;

    // clang-format off
    realtype nH = d_udata->nH;
    realtype Tgas = d_udata->Tgas;
    
    realtype mu = d_udata->mu;
    realtype gamma = d_udata->gamma;
        
    if (mu < 0) mu = GetMu(y);
    if (gamma < 0) gamma = GetGamma(y);
    // clang-format on

    for (int cur = tidx; cur < nsystem; cur += gs) {
        int yistart            = cur * NEQUATIONS;
        realtype *y_cur        = y + yistart;
        realtype k[NREACTIONS] = {0.0};
        NaunetData *udata      = &d_udata[cur];

        EvalRates(k, y_cur, udata);

        // clang-format off
        ydot[yistart + IDX_DI] = 0.0 - k[29]*y_cur[IDX_HII]*y_cur[IDX_DI] +
            k[30]*y_cur[IDX_HI]*y_cur[IDX_DII] -
            k[33]*y_cur[IDX_H2I]*y_cur[IDX_DI] -
            k[34]*y_cur[IDX_H2I]*y_cur[IDX_DI] +
            k[35]*y_cur[IDX_HDI]*y_cur[IDX_HI] -
            k[36]*y_cur[IDX_DI]*y_cur[IDX_HM] +
            k[37]*y_cur[IDX_DII]*y_cur[IDX_eM];
        ydot[yistart + IDX_DII] = 0.0 + k[29]*y_cur[IDX_HII]*y_cur[IDX_DI] -
            k[30]*y_cur[IDX_HI]*y_cur[IDX_DII] -
            k[31]*y_cur[IDX_H2I]*y_cur[IDX_DII] +
            k[32]*y_cur[IDX_HDI]*y_cur[IDX_HII] -
            k[37]*y_cur[IDX_DII]*y_cur[IDX_eM];
        ydot[yistart + IDX_GRAINI] = 0.0 + k[1]*y_cur[IDX_HII]*y_cur[IDX_eM]
            + k[2]*y_cur[IDX_HII]*y_cur[IDX_eM] +
            k[4]*y_cur[IDX_HeII]*y_cur[IDX_eM] +
            k[5]*y_cur[IDX_HeII]*y_cur[IDX_eM] +
            k[7]*y_cur[IDX_HeIII]*y_cur[IDX_eM] +
            k[8]*y_cur[IDX_HI]*y_cur[IDX_eM] +
            k[11]*y_cur[IDX_HI]*y_cur[IDX_HII] +
            k[12]*y_cur[IDX_HI]*y_cur[IDX_HII];
        ydot[yistart + IDX_HI] = 0.0 - k[0]*y_cur[IDX_HI]*y_cur[IDX_eM] +
            k[1]*y_cur[IDX_HII]*y_cur[IDX_eM] +
            k[2]*y_cur[IDX_HII]*y_cur[IDX_eM] - k[8]*y_cur[IDX_HI]*y_cur[IDX_eM]
            - k[9]*y_cur[IDX_HM]*y_cur[IDX_HI] -
            k[10]*y_cur[IDX_HM]*y_cur[IDX_HI] -
            k[11]*y_cur[IDX_HI]*y_cur[IDX_HII] -
            k[12]*y_cur[IDX_HI]*y_cur[IDX_HII] -
            k[13]*y_cur[IDX_H2II]*y_cur[IDX_HI] +
            k[14]*y_cur[IDX_H2I]*y_cur[IDX_HII] +
            k[15]*y_cur[IDX_H2I]*y_cur[IDX_eM] +
            k[15]*y_cur[IDX_H2I]*y_cur[IDX_eM] -
            k[16]*y_cur[IDX_H2I]*y_cur[IDX_HI] +
            k[16]*y_cur[IDX_H2I]*y_cur[IDX_HI] +
            k[16]*y_cur[IDX_H2I]*y_cur[IDX_HI] +
            k[16]*y_cur[IDX_H2I]*y_cur[IDX_HI] +
            k[17]*y_cur[IDX_HM]*y_cur[IDX_eM] -
            k[18]*y_cur[IDX_HM]*y_cur[IDX_HI] +
            k[18]*y_cur[IDX_HM]*y_cur[IDX_HI] +
            k[18]*y_cur[IDX_HM]*y_cur[IDX_HI] -
            k[19]*y_cur[IDX_HM]*y_cur[IDX_HI] +
            k[19]*y_cur[IDX_HM]*y_cur[IDX_HI] +
            k[19]*y_cur[IDX_HM]*y_cur[IDX_HI] +
            k[20]*y_cur[IDX_HM]*y_cur[IDX_HII] +
            k[20]*y_cur[IDX_HM]*y_cur[IDX_HII] +
            k[22]*y_cur[IDX_H2II]*y_cur[IDX_eM] +
            k[22]*y_cur[IDX_H2II]*y_cur[IDX_eM] +
            k[23]*y_cur[IDX_H2II]*y_cur[IDX_eM] +
            k[23]*y_cur[IDX_H2II]*y_cur[IDX_eM] +
            k[24]*y_cur[IDX_H2II]*y_cur[IDX_HM] -
            k[25]*y_cur[IDX_HI]*y_cur[IDX_HI]*y_cur[IDX_HI] -
            k[25]*y_cur[IDX_HI]*y_cur[IDX_HI]*y_cur[IDX_HI] -
            k[25]*y_cur[IDX_HI]*y_cur[IDX_HI]*y_cur[IDX_HI] +
            k[25]*y_cur[IDX_HI]*y_cur[IDX_HI]*y_cur[IDX_HI] -
            k[26]*y_cur[IDX_HI]*y_cur[IDX_HI]*y_cur[IDX_HI] -
            k[26]*y_cur[IDX_HI]*y_cur[IDX_HI]*y_cur[IDX_HI] -
            k[26]*y_cur[IDX_HI]*y_cur[IDX_HI]*y_cur[IDX_HI] +
            k[26]*y_cur[IDX_HI]*y_cur[IDX_HI]*y_cur[IDX_HI] -
            k[27]*y_cur[IDX_H2I]*y_cur[IDX_HI]*y_cur[IDX_HI] -
            k[27]*y_cur[IDX_H2I]*y_cur[IDX_HI]*y_cur[IDX_HI] -
            k[28]*y_cur[IDX_H2I]*y_cur[IDX_HI]*y_cur[IDX_HI] -
            k[28]*y_cur[IDX_H2I]*y_cur[IDX_HI]*y_cur[IDX_HI] +
            k[29]*y_cur[IDX_HII]*y_cur[IDX_DI] -
            k[30]*y_cur[IDX_HI]*y_cur[IDX_DII] +
            k[33]*y_cur[IDX_H2I]*y_cur[IDX_DI] +
            k[34]*y_cur[IDX_H2I]*y_cur[IDX_DI] -
            k[35]*y_cur[IDX_HDI]*y_cur[IDX_HI];
        ydot[yistart + IDX_HII] = 0.0 + k[0]*y_cur[IDX_HI]*y_cur[IDX_eM] -
            k[1]*y_cur[IDX_HII]*y_cur[IDX_eM] -
            k[2]*y_cur[IDX_HII]*y_cur[IDX_eM] -
            k[11]*y_cur[IDX_HI]*y_cur[IDX_HII] -
            k[12]*y_cur[IDX_HI]*y_cur[IDX_HII] +
            k[13]*y_cur[IDX_H2II]*y_cur[IDX_HI] -
            k[14]*y_cur[IDX_H2I]*y_cur[IDX_HII] -
            k[20]*y_cur[IDX_HM]*y_cur[IDX_HII] -
            k[21]*y_cur[IDX_HM]*y_cur[IDX_HII] -
            k[29]*y_cur[IDX_HII]*y_cur[IDX_DI] +
            k[30]*y_cur[IDX_HI]*y_cur[IDX_DII] +
            k[31]*y_cur[IDX_H2I]*y_cur[IDX_DII] -
            k[32]*y_cur[IDX_HDI]*y_cur[IDX_HII];
        ydot[yistart + IDX_HM] = 0.0 + k[8]*y_cur[IDX_HI]*y_cur[IDX_eM] -
            k[9]*y_cur[IDX_HM]*y_cur[IDX_HI] - k[10]*y_cur[IDX_HM]*y_cur[IDX_HI]
            - k[17]*y_cur[IDX_HM]*y_cur[IDX_eM] -
            k[18]*y_cur[IDX_HM]*y_cur[IDX_HI] -
            k[19]*y_cur[IDX_HM]*y_cur[IDX_HI] -
            k[20]*y_cur[IDX_HM]*y_cur[IDX_HII] -
            k[21]*y_cur[IDX_HM]*y_cur[IDX_HII] -
            k[24]*y_cur[IDX_H2II]*y_cur[IDX_HM] -
            k[36]*y_cur[IDX_DI]*y_cur[IDX_HM];
        ydot[yistart + IDX_H2I] = 0.0 + k[9]*y_cur[IDX_HM]*y_cur[IDX_HI] +
            k[10]*y_cur[IDX_HM]*y_cur[IDX_HI] +
            k[13]*y_cur[IDX_H2II]*y_cur[IDX_HI] -
            k[14]*y_cur[IDX_H2I]*y_cur[IDX_HII] -
            k[15]*y_cur[IDX_H2I]*y_cur[IDX_eM] -
            k[16]*y_cur[IDX_H2I]*y_cur[IDX_HI] +
            k[24]*y_cur[IDX_H2II]*y_cur[IDX_HM] +
            k[25]*y_cur[IDX_HI]*y_cur[IDX_HI]*y_cur[IDX_HI] +
            k[26]*y_cur[IDX_HI]*y_cur[IDX_HI]*y_cur[IDX_HI] -
            k[27]*y_cur[IDX_H2I]*y_cur[IDX_HI]*y_cur[IDX_HI] +
            k[27]*y_cur[IDX_H2I]*y_cur[IDX_HI]*y_cur[IDX_HI] +
            k[27]*y_cur[IDX_H2I]*y_cur[IDX_HI]*y_cur[IDX_HI] -
            k[28]*y_cur[IDX_H2I]*y_cur[IDX_HI]*y_cur[IDX_HI] +
            k[28]*y_cur[IDX_H2I]*y_cur[IDX_HI]*y_cur[IDX_HI] +
            k[28]*y_cur[IDX_H2I]*y_cur[IDX_HI]*y_cur[IDX_HI] -
            k[31]*y_cur[IDX_H2I]*y_cur[IDX_DII] +
            k[32]*y_cur[IDX_HDI]*y_cur[IDX_HII] -
            k[33]*y_cur[IDX_H2I]*y_cur[IDX_DI] -
            k[34]*y_cur[IDX_H2I]*y_cur[IDX_DI] +
            k[35]*y_cur[IDX_HDI]*y_cur[IDX_HI];
        ydot[yistart + IDX_H2II] = 0.0 + k[11]*y_cur[IDX_HI]*y_cur[IDX_HII]
            + k[12]*y_cur[IDX_HI]*y_cur[IDX_HII] -
            k[13]*y_cur[IDX_H2II]*y_cur[IDX_HI] +
            k[14]*y_cur[IDX_H2I]*y_cur[IDX_HII] +
            k[21]*y_cur[IDX_HM]*y_cur[IDX_HII] -
            k[22]*y_cur[IDX_H2II]*y_cur[IDX_eM] -
            k[23]*y_cur[IDX_H2II]*y_cur[IDX_eM] -
            k[24]*y_cur[IDX_H2II]*y_cur[IDX_HM];
        ydot[yistart + IDX_HDI] = 0.0 + k[31]*y_cur[IDX_H2I]*y_cur[IDX_DII]
            - k[32]*y_cur[IDX_HDI]*y_cur[IDX_HII] +
            k[33]*y_cur[IDX_H2I]*y_cur[IDX_DI] +
            k[34]*y_cur[IDX_H2I]*y_cur[IDX_DI] -
            k[35]*y_cur[IDX_HDI]*y_cur[IDX_HI] +
            k[36]*y_cur[IDX_DI]*y_cur[IDX_HM];
        ydot[yistart + IDX_HeI] = 0.0 - k[3]*y_cur[IDX_HeI]*y_cur[IDX_eM] +
            k[4]*y_cur[IDX_HeII]*y_cur[IDX_eM] +
            k[5]*y_cur[IDX_HeII]*y_cur[IDX_eM];
        ydot[yistart + IDX_HeII] = 0.0 + k[3]*y_cur[IDX_HeI]*y_cur[IDX_eM] -
            k[4]*y_cur[IDX_HeII]*y_cur[IDX_eM] -
            k[5]*y_cur[IDX_HeII]*y_cur[IDX_eM] -
            k[6]*y_cur[IDX_HeII]*y_cur[IDX_eM] +
            k[7]*y_cur[IDX_HeIII]*y_cur[IDX_eM];
        ydot[yistart + IDX_HeIII] = 0.0 + k[6]*y_cur[IDX_HeII]*y_cur[IDX_eM]
            - k[7]*y_cur[IDX_HeIII]*y_cur[IDX_eM];
        ydot[yistart + IDX_eM] = 0.0 - k[0]*y_cur[IDX_HI]*y_cur[IDX_eM] +
            k[0]*y_cur[IDX_HI]*y_cur[IDX_eM] + k[0]*y_cur[IDX_HI]*y_cur[IDX_eM]
            - k[1]*y_cur[IDX_HII]*y_cur[IDX_eM] -
            k[2]*y_cur[IDX_HII]*y_cur[IDX_eM] -
            k[3]*y_cur[IDX_HeI]*y_cur[IDX_eM] +
            k[3]*y_cur[IDX_HeI]*y_cur[IDX_eM] +
            k[3]*y_cur[IDX_HeI]*y_cur[IDX_eM] -
            k[4]*y_cur[IDX_HeII]*y_cur[IDX_eM] -
            k[5]*y_cur[IDX_HeII]*y_cur[IDX_eM] -
            k[6]*y_cur[IDX_HeII]*y_cur[IDX_eM] +
            k[6]*y_cur[IDX_HeII]*y_cur[IDX_eM] +
            k[6]*y_cur[IDX_HeII]*y_cur[IDX_eM] -
            k[7]*y_cur[IDX_HeIII]*y_cur[IDX_eM] -
            k[8]*y_cur[IDX_HI]*y_cur[IDX_eM] + k[9]*y_cur[IDX_HM]*y_cur[IDX_HI]
            + k[10]*y_cur[IDX_HM]*y_cur[IDX_HI] -
            k[15]*y_cur[IDX_H2I]*y_cur[IDX_eM] +
            k[15]*y_cur[IDX_H2I]*y_cur[IDX_eM] -
            k[17]*y_cur[IDX_HM]*y_cur[IDX_eM] +
            k[17]*y_cur[IDX_HM]*y_cur[IDX_eM] +
            k[17]*y_cur[IDX_HM]*y_cur[IDX_eM] +
            k[18]*y_cur[IDX_HM]*y_cur[IDX_HI] +
            k[19]*y_cur[IDX_HM]*y_cur[IDX_HI] +
            k[21]*y_cur[IDX_HM]*y_cur[IDX_HII] -
            k[22]*y_cur[IDX_H2II]*y_cur[IDX_eM] -
            k[23]*y_cur[IDX_H2II]*y_cur[IDX_eM] +
            k[36]*y_cur[IDX_DI]*y_cur[IDX_HM] -
            k[37]*y_cur[IDX_DII]*y_cur[IDX_eM];
        ydot[yistart + IDX_TGAS] = (gamma - 1.0) * ( 0.0 - 1.27e-21 *
            sqrt(y_cur[IDX_TGAS]) / (1.0 + sqrt(y_cur[IDX_TGAS]/1e5)) *
            exp(-1.578091e5/y_cur[IDX_TGAS]) * y_cur[IDX_HI]*y_cur[IDX_eM] -
            9.38e-22 * sqrt(y_cur[IDX_TGAS]) / (1.0 + sqrt(y_cur[IDX_TGAS]/1e5))
            * exp(-2.853354e5/y_cur[IDX_TGAS]) * y_cur[IDX_HeI]*y_cur[IDX_eM] -
            4.95e-22 * sqrt(y_cur[IDX_TGAS]) / (1.0 + sqrt(y_cur[IDX_TGAS]/1e5))
            * exp(-6.31515e5/y_cur[IDX_TGAS]) * y_cur[IDX_HeII]*y_cur[IDX_eM] -
            5.01e-27 * pow(y_cur[IDX_TGAS], -0.1687) / (1.0 +
            sqrt(y_cur[IDX_TGAS]/1e5)) * exp(-5.5338e4/y_cur[IDX_TGAS]) *
            y_cur[IDX_HeII]*y_cur[IDX_eM]*y_cur[IDX_eM] - 8.7e-27 *
            sqrt(y_cur[IDX_TGAS]) * pow(y_cur[IDX_TGAS]/1e3, -0.2) /
            (1.0+pow(y_cur[IDX_TGAS]/1e6, 0.7)) * y_cur[IDX_HII]*y_cur[IDX_eM] -
            1.24e-13 * pow(y_cur[IDX_TGAS], -1.5) * exp(-4.7e5/y_cur[IDX_TGAS])
            * (1.0+0.3*exp(-9.4e4/y_cur[IDX_TGAS])) *
            y_cur[IDX_HeII]*y_cur[IDX_eM] - 1.55e-26 * pow(y_cur[IDX_TGAS],
            0.3647) * y_cur[IDX_HeII]*y_cur[IDX_eM] - 3.48e-26 *
            sqrt(y_cur[IDX_TGAS]) * pow(y_cur[IDX_TGAS]/1e3, -0.2) /
            (1.0+pow(y_cur[IDX_TGAS]/1e6, 0.7)) * y_cur[IDX_HeIII]*y_cur[IDX_eM]
            - 9.1e-27 * pow(y_cur[IDX_TGAS], -0.1687) /
            (1.0+sqrt(y_cur[IDX_TGAS]/1e5)) * exp(-1.3179e4/y_cur[IDX_TGAS]) *
            y_cur[IDX_HI]*y_cur[IDX_eM]*y_cur[IDX_eM] - 5.54e-17 *
            pow(y_cur[IDX_TGAS], -.0397) / (1.0+sqrt(y_cur[IDX_TGAS]/1e5))
            *exp(-4.73638e5/y_cur[IDX_TGAS]) * y_cur[IDX_HeII]*y_cur[IDX_eM] -
            5.54e-17 * pow(y_cur[IDX_TGAS], -.0397) /
            (1.0+sqrt(y_cur[IDX_TGAS]/1e5)) *exp(-4.73638e5/y_cur[IDX_TGAS]) *
            y_cur[IDX_HeII]*y_cur[IDX_eM] ) / kerg / GetNumDens(y);
        
                // clang-format on
    }
}

__global__ void JacKernel(realtype *y, realtype *data, NaunetData *d_udata,
                          int nsystem) {
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gs   = blockDim.x * gridDim.x;

    for (int cur = tidx; cur < nsystem; cur += gs) {
        int yistart            = cur * NEQUATIONS;
        int jistart            = cur * NNZ;
        realtype *y_cur        = y + yistart;
        realtype k[NREACTIONS] = {0.0};
        NaunetData *udata      = &d_udata[cur];

        // clang-format off
        realtype mu = udata->mu;
        realtype gamma = udata->gamma;
                
        if (mu < 0) mu = GetMu(y);
        if (gamma < 0) gamma = GetGamma(y);
        // clang-format on

        EvalRates(k, y_cur, udata);

        // clang-format off
        data[jistart + 0] = 0.0 - k[29]*y_cur[IDX_HII] - k[33]*y_cur[IDX_H2I] - k[34]*y_cur[IDX_H2I] - k[36]*y_cur[IDX_HM];
        data[jistart + 1] = 0.0 + k[30]*y_cur[IDX_HI] + k[37]*y_cur[IDX_eM];
        data[jistart + 2] = 0.0 + k[30]*y_cur[IDX_DII] + k[35]*y_cur[IDX_HDI];
        data[jistart + 3] = 0.0 - k[29]*y_cur[IDX_DI];
        data[jistart + 4] = 0.0 - k[36]*y_cur[IDX_DI];
        data[jistart + 5] = 0.0 - k[33]*y_cur[IDX_DI] - k[34]*y_cur[IDX_DI];
        data[jistart + 6] = 0.0 + k[35]*y_cur[IDX_HI];
        data[jistart + 7] = 0.0 + k[37]*y_cur[IDX_DII];
        data[jistart + 8] = 0.0 + k[29]*y_cur[IDX_HII];
        data[jistart + 9] = 0.0 - k[30]*y_cur[IDX_HI] - k[31]*y_cur[IDX_H2I] - k[37]*y_cur[IDX_eM];
        data[jistart + 10] = 0.0 - k[30]*y_cur[IDX_DII];
        data[jistart + 11] = 0.0 + k[29]*y_cur[IDX_DI] + k[32]*y_cur[IDX_HDI];
        data[jistart + 12] = 0.0 - k[31]*y_cur[IDX_DII];
        data[jistart + 13] = 0.0 + k[32]*y_cur[IDX_HII];
        data[jistart + 14] = 0.0 - k[37]*y_cur[IDX_DII];
        data[jistart + 15] = 0.0 + k[8]*y_cur[IDX_eM] + k[11]*y_cur[IDX_HII] + k[12]*y_cur[IDX_HII];
        data[jistart + 16] = 0.0 + k[1]*y_cur[IDX_eM] + k[2]*y_cur[IDX_eM] + k[11]*y_cur[IDX_HI] + k[12]*y_cur[IDX_HI];
        data[jistart + 17] = 0.0 + k[4]*y_cur[IDX_eM] + k[5]*y_cur[IDX_eM];
        data[jistart + 18] = 0.0 + k[7]*y_cur[IDX_eM];
        data[jistart + 19] = 0.0 + k[1]*y_cur[IDX_HII] + k[2]*y_cur[IDX_HII] + k[4]*y_cur[IDX_HeII] + k[5]*y_cur[IDX_HeII] + k[7]*y_cur[IDX_HeIII] + k[8]*y_cur[IDX_HI];
        data[jistart + 20] = 0.0 + k[29]*y_cur[IDX_HII] + k[33]*y_cur[IDX_H2I] + k[34]*y_cur[IDX_H2I];
        data[jistart + 21] = 0.0 - k[30]*y_cur[IDX_HI];
        data[jistart + 22] = 0.0 - k[0]*y_cur[IDX_eM] - k[8]*y_cur[IDX_eM] - k[9]*y_cur[IDX_HM] - k[10]*y_cur[IDX_HM] - k[11]*y_cur[IDX_HII] - k[12]*y_cur[IDX_HII] - k[13]*y_cur[IDX_H2II] - k[16]*y_cur[IDX_H2I] + k[16]*y_cur[IDX_H2I] + k[16]*y_cur[IDX_H2I] + k[16]*y_cur[IDX_H2I] - k[18]*y_cur[IDX_HM] + k[18]*y_cur[IDX_HM] + k[18]*y_cur[IDX_HM] - k[19]*y_cur[IDX_HM] + k[19]*y_cur[IDX_HM] + k[19]*y_cur[IDX_HM] - k[25]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[25]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[25]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[25]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[25]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[25]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[25]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[25]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[25]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[25]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[25]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[25]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[26]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[26]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[26]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[26]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[26]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[26]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[26]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[26]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[26]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[26]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[26]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[26]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[27]*y_cur[IDX_H2I]*y_cur[IDX_HI] - k[27]*y_cur[IDX_H2I]*y_cur[IDX_HI] - k[27]*y_cur[IDX_H2I]*y_cur[IDX_HI] - k[27]*y_cur[IDX_H2I]*y_cur[IDX_HI] - k[28]*y_cur[IDX_H2I]*y_cur[IDX_HI] - k[28]*y_cur[IDX_H2I]*y_cur[IDX_HI] - k[28]*y_cur[IDX_H2I]*y_cur[IDX_HI] - k[28]*y_cur[IDX_H2I]*y_cur[IDX_HI] - k[30]*y_cur[IDX_DII] - k[35]*y_cur[IDX_HDI];
        data[jistart + 23] = 0.0 + k[1]*y_cur[IDX_eM] + k[2]*y_cur[IDX_eM] - k[11]*y_cur[IDX_HI] - k[12]*y_cur[IDX_HI] + k[14]*y_cur[IDX_H2I] + k[20]*y_cur[IDX_HM] + k[20]*y_cur[IDX_HM] + k[29]*y_cur[IDX_DI];
        data[jistart + 24] = 0.0 - k[9]*y_cur[IDX_HI] - k[10]*y_cur[IDX_HI] + k[17]*y_cur[IDX_eM] - k[18]*y_cur[IDX_HI] + k[18]*y_cur[IDX_HI] + k[18]*y_cur[IDX_HI] - k[19]*y_cur[IDX_HI] + k[19]*y_cur[IDX_HI] + k[19]*y_cur[IDX_HI] + k[20]*y_cur[IDX_HII] + k[20]*y_cur[IDX_HII] + k[24]*y_cur[IDX_H2II];
        data[jistart + 25] = 0.0 + k[14]*y_cur[IDX_HII] + k[15]*y_cur[IDX_eM] + k[15]*y_cur[IDX_eM] - k[16]*y_cur[IDX_HI] + k[16]*y_cur[IDX_HI] + k[16]*y_cur[IDX_HI] + k[16]*y_cur[IDX_HI] - k[27]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[27]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[28]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[28]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[33]*y_cur[IDX_DI] + k[34]*y_cur[IDX_DI];
        data[jistart + 26] = 0.0 - k[13]*y_cur[IDX_HI] + k[22]*y_cur[IDX_eM] + k[22]*y_cur[IDX_eM] + k[23]*y_cur[IDX_eM] + k[23]*y_cur[IDX_eM] + k[24]*y_cur[IDX_HM];
        data[jistart + 27] = 0.0 - k[35]*y_cur[IDX_HI];
        data[jistart + 28] = 0.0 - k[0]*y_cur[IDX_HI] + k[1]*y_cur[IDX_HII] + k[2]*y_cur[IDX_HII] - k[8]*y_cur[IDX_HI] + k[15]*y_cur[IDX_H2I] + k[15]*y_cur[IDX_H2I] + k[17]*y_cur[IDX_HM] + k[22]*y_cur[IDX_H2II] + k[22]*y_cur[IDX_H2II] + k[23]*y_cur[IDX_H2II] + k[23]*y_cur[IDX_H2II];
        data[jistart + 29] = 0.0 - k[29]*y_cur[IDX_HII];
        data[jistart + 30] = 0.0 + k[30]*y_cur[IDX_HI] + k[31]*y_cur[IDX_H2I];
        data[jistart + 31] = 0.0 + k[0]*y_cur[IDX_eM] - k[11]*y_cur[IDX_HII] - k[12]*y_cur[IDX_HII] + k[13]*y_cur[IDX_H2II] + k[30]*y_cur[IDX_DII];
        data[jistart + 32] = 0.0 - k[1]*y_cur[IDX_eM] - k[2]*y_cur[IDX_eM] - k[11]*y_cur[IDX_HI] - k[12]*y_cur[IDX_HI] - k[14]*y_cur[IDX_H2I] - k[20]*y_cur[IDX_HM] - k[21]*y_cur[IDX_HM] - k[29]*y_cur[IDX_DI] - k[32]*y_cur[IDX_HDI];
        data[jistart + 33] = 0.0 - k[20]*y_cur[IDX_HII] - k[21]*y_cur[IDX_HII];
        data[jistart + 34] = 0.0 - k[14]*y_cur[IDX_HII] + k[31]*y_cur[IDX_DII];
        data[jistart + 35] = 0.0 + k[13]*y_cur[IDX_HI];
        data[jistart + 36] = 0.0 - k[32]*y_cur[IDX_HII];
        data[jistart + 37] = 0.0 + k[0]*y_cur[IDX_HI] - k[1]*y_cur[IDX_HII] - k[2]*y_cur[IDX_HII];
        data[jistart + 38] = 0.0 - k[36]*y_cur[IDX_HM];
        data[jistart + 39] = 0.0 + k[8]*y_cur[IDX_eM] - k[9]*y_cur[IDX_HM] - k[10]*y_cur[IDX_HM] - k[18]*y_cur[IDX_HM] - k[19]*y_cur[IDX_HM];
        data[jistart + 40] = 0.0 - k[20]*y_cur[IDX_HM] - k[21]*y_cur[IDX_HM];
        data[jistart + 41] = 0.0 - k[9]*y_cur[IDX_HI] - k[10]*y_cur[IDX_HI] - k[17]*y_cur[IDX_eM] - k[18]*y_cur[IDX_HI] - k[19]*y_cur[IDX_HI] - k[20]*y_cur[IDX_HII] - k[21]*y_cur[IDX_HII] - k[24]*y_cur[IDX_H2II] - k[36]*y_cur[IDX_DI];
        data[jistart + 42] = 0.0 - k[24]*y_cur[IDX_HM];
        data[jistart + 43] = 0.0 + k[8]*y_cur[IDX_HI] - k[17]*y_cur[IDX_HM];
        data[jistart + 44] = 0.0 - k[33]*y_cur[IDX_H2I] - k[34]*y_cur[IDX_H2I];
        data[jistart + 45] = 0.0 - k[31]*y_cur[IDX_H2I];
        data[jistart + 46] = 0.0 + k[9]*y_cur[IDX_HM] + k[10]*y_cur[IDX_HM] + k[13]*y_cur[IDX_H2II] - k[16]*y_cur[IDX_H2I] + k[25]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[25]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[25]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[26]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[26]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[26]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[27]*y_cur[IDX_H2I]*y_cur[IDX_HI] - k[27]*y_cur[IDX_H2I]*y_cur[IDX_HI] + k[27]*y_cur[IDX_H2I]*y_cur[IDX_HI] + k[27]*y_cur[IDX_H2I]*y_cur[IDX_HI] + k[27]*y_cur[IDX_H2I]*y_cur[IDX_HI] + k[27]*y_cur[IDX_H2I]*y_cur[IDX_HI] - k[28]*y_cur[IDX_H2I]*y_cur[IDX_HI] - k[28]*y_cur[IDX_H2I]*y_cur[IDX_HI] + k[28]*y_cur[IDX_H2I]*y_cur[IDX_HI] + k[28]*y_cur[IDX_H2I]*y_cur[IDX_HI] + k[28]*y_cur[IDX_H2I]*y_cur[IDX_HI] + k[28]*y_cur[IDX_H2I]*y_cur[IDX_HI] + k[35]*y_cur[IDX_HDI];
        data[jistart + 47] = 0.0 - k[14]*y_cur[IDX_H2I] + k[32]*y_cur[IDX_HDI];
        data[jistart + 48] = 0.0 + k[9]*y_cur[IDX_HI] + k[10]*y_cur[IDX_HI] + k[24]*y_cur[IDX_H2II];
        data[jistart + 49] = 0.0 - k[14]*y_cur[IDX_HII] - k[15]*y_cur[IDX_eM] - k[16]*y_cur[IDX_HI] - k[27]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[27]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[27]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[28]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[28]*y_cur[IDX_HI]*y_cur[IDX_HI] + k[28]*y_cur[IDX_HI]*y_cur[IDX_HI] - k[31]*y_cur[IDX_DII] - k[33]*y_cur[IDX_DI] - k[34]*y_cur[IDX_DI];
        data[jistart + 50] = 0.0 + k[13]*y_cur[IDX_HI] + k[24]*y_cur[IDX_HM];
        data[jistart + 51] = 0.0 + k[32]*y_cur[IDX_HII] + k[35]*y_cur[IDX_HI];
        data[jistart + 52] = 0.0 - k[15]*y_cur[IDX_H2I];
        data[jistart + 53] = 0.0 + k[11]*y_cur[IDX_HII] + k[12]*y_cur[IDX_HII] - k[13]*y_cur[IDX_H2II];
        data[jistart + 54] = 0.0 + k[11]*y_cur[IDX_HI] + k[12]*y_cur[IDX_HI] + k[14]*y_cur[IDX_H2I] + k[21]*y_cur[IDX_HM];
        data[jistart + 55] = 0.0 + k[21]*y_cur[IDX_HII] - k[24]*y_cur[IDX_H2II];
        data[jistart + 56] = 0.0 + k[14]*y_cur[IDX_HII];
        data[jistart + 57] = 0.0 - k[13]*y_cur[IDX_HI] - k[22]*y_cur[IDX_eM] - k[23]*y_cur[IDX_eM] - k[24]*y_cur[IDX_HM];
        data[jistart + 58] = 0.0 - k[22]*y_cur[IDX_H2II] - k[23]*y_cur[IDX_H2II];
        data[jistart + 59] = 0.0 + k[33]*y_cur[IDX_H2I] + k[34]*y_cur[IDX_H2I] + k[36]*y_cur[IDX_HM];
        data[jistart + 60] = 0.0 + k[31]*y_cur[IDX_H2I];
        data[jistart + 61] = 0.0 - k[35]*y_cur[IDX_HDI];
        data[jistart + 62] = 0.0 - k[32]*y_cur[IDX_HDI];
        data[jistart + 63] = 0.0 + k[36]*y_cur[IDX_DI];
        data[jistart + 64] = 0.0 + k[31]*y_cur[IDX_DII] + k[33]*y_cur[IDX_DI] + k[34]*y_cur[IDX_DI];
        data[jistart + 65] = 0.0 - k[32]*y_cur[IDX_HII] - k[35]*y_cur[IDX_HI];
        data[jistart + 66] = 0.0 - k[3]*y_cur[IDX_eM];
        data[jistart + 67] = 0.0 + k[4]*y_cur[IDX_eM] + k[5]*y_cur[IDX_eM];
        data[jistart + 68] = 0.0 - k[3]*y_cur[IDX_HeI] + k[4]*y_cur[IDX_HeII] + k[5]*y_cur[IDX_HeII];
        data[jistart + 69] = 0.0 + k[3]*y_cur[IDX_eM];
        data[jistart + 70] = 0.0 - k[4]*y_cur[IDX_eM] - k[5]*y_cur[IDX_eM] - k[6]*y_cur[IDX_eM];
        data[jistart + 71] = 0.0 + k[7]*y_cur[IDX_eM];
        data[jistart + 72] = 0.0 + k[3]*y_cur[IDX_HeI] - k[4]*y_cur[IDX_HeII] - k[5]*y_cur[IDX_HeII] - k[6]*y_cur[IDX_HeII] + k[7]*y_cur[IDX_HeIII];
        data[jistart + 73] = 0.0 + k[6]*y_cur[IDX_eM];
        data[jistart + 74] = 0.0 - k[7]*y_cur[IDX_eM];
        data[jistart + 75] = 0.0 + k[6]*y_cur[IDX_HeII] - k[7]*y_cur[IDX_HeIII];
        data[jistart + 76] = 0.0 + k[36]*y_cur[IDX_HM];
        data[jistart + 77] = 0.0 - k[37]*y_cur[IDX_eM];
        data[jistart + 78] = 0.0 - k[0]*y_cur[IDX_eM] + k[0]*y_cur[IDX_eM] + k[0]*y_cur[IDX_eM] - k[8]*y_cur[IDX_eM] + k[9]*y_cur[IDX_HM] + k[10]*y_cur[IDX_HM] + k[18]*y_cur[IDX_HM] + k[19]*y_cur[IDX_HM];
        data[jistart + 79] = 0.0 - k[1]*y_cur[IDX_eM] - k[2]*y_cur[IDX_eM] + k[21]*y_cur[IDX_HM];
        data[jistart + 80] = 0.0 + k[9]*y_cur[IDX_HI] + k[10]*y_cur[IDX_HI] - k[17]*y_cur[IDX_eM] + k[17]*y_cur[IDX_eM] + k[17]*y_cur[IDX_eM] + k[18]*y_cur[IDX_HI] + k[19]*y_cur[IDX_HI] + k[21]*y_cur[IDX_HII] + k[36]*y_cur[IDX_DI];
        data[jistart + 81] = 0.0 - k[15]*y_cur[IDX_eM] + k[15]*y_cur[IDX_eM];
        data[jistart + 82] = 0.0 - k[22]*y_cur[IDX_eM] - k[23]*y_cur[IDX_eM];
        data[jistart + 83] = 0.0 - k[3]*y_cur[IDX_eM] + k[3]*y_cur[IDX_eM] + k[3]*y_cur[IDX_eM];
        data[jistart + 84] = 0.0 - k[4]*y_cur[IDX_eM] - k[5]*y_cur[IDX_eM] - k[6]*y_cur[IDX_eM] + k[6]*y_cur[IDX_eM] + k[6]*y_cur[IDX_eM];
        data[jistart + 85] = 0.0 - k[7]*y_cur[IDX_eM];
        data[jistart + 86] = 0.0 - k[0]*y_cur[IDX_HI] + k[0]*y_cur[IDX_HI] + k[0]*y_cur[IDX_HI] - k[1]*y_cur[IDX_HII] - k[2]*y_cur[IDX_HII] - k[3]*y_cur[IDX_HeI] + k[3]*y_cur[IDX_HeI] + k[3]*y_cur[IDX_HeI] - k[4]*y_cur[IDX_HeII] - k[5]*y_cur[IDX_HeII] - k[6]*y_cur[IDX_HeII] + k[6]*y_cur[IDX_HeII] + k[6]*y_cur[IDX_HeII] - k[7]*y_cur[IDX_HeIII] - k[8]*y_cur[IDX_HI] - k[15]*y_cur[IDX_H2I] + k[15]*y_cur[IDX_H2I] - k[17]*y_cur[IDX_HM] + k[17]*y_cur[IDX_HM] + k[17]*y_cur[IDX_HM] - k[22]*y_cur[IDX_H2II] - k[23]*y_cur[IDX_H2II] - k[37]*y_cur[IDX_DII];
        data[jistart + 87] = 0.0 - 1.27e-21 * sqrt(y_cur[IDX_TGAS]) / (1.0 + sqrt(y_cur[IDX_TGAS]/1e5)) * exp(-1.578091e5/y_cur[IDX_TGAS])*y_cur[IDX_eM] - 9.1e-27 * pow(y_cur[IDX_TGAS], -0.1687) / (1.0+sqrt(y_cur[IDX_TGAS]/1e5)) * exp(-1.3179e4/y_cur[IDX_TGAS])*y_cur[IDX_eM]*y_cur[IDX_eM];
        data[jistart + 88] = 0.0 - 8.7e-27 * sqrt(y_cur[IDX_TGAS]) * pow(y_cur[IDX_TGAS]/1e3, -0.2) / (1.0+pow(y_cur[IDX_TGAS]/1e6, 0.7))*y_cur[IDX_eM];
        data[jistart + 89] = 0.0 - 9.38e-22 * sqrt(y_cur[IDX_TGAS]) / (1.0 + sqrt(y_cur[IDX_TGAS]/1e5)) * exp(-2.853354e5/y_cur[IDX_TGAS])*y_cur[IDX_eM];
        data[jistart + 90] = 0.0 - 4.95e-22 * sqrt(y_cur[IDX_TGAS]) / (1.0 + sqrt(y_cur[IDX_TGAS]/1e5)) * exp(-6.31515e5/y_cur[IDX_TGAS])*y_cur[IDX_eM] - 5.01e-27 * pow(y_cur[IDX_TGAS], -0.1687) / (1.0 + sqrt(y_cur[IDX_TGAS]/1e5)) * exp(-5.5338e4/y_cur[IDX_TGAS])*y_cur[IDX_eM]*y_cur[IDX_eM] - 1.24e-13 * pow(y_cur[IDX_TGAS], -1.5) * exp(-4.7e5/y_cur[IDX_TGAS]) * (1.0+0.3*exp(-9.4e4/y_cur[IDX_TGAS]))*y_cur[IDX_eM] - 1.55e-26 * pow(y_cur[IDX_TGAS], 0.3647)*y_cur[IDX_eM] - 5.54e-17 * pow(y_cur[IDX_TGAS], -.0397) / (1.0+sqrt(y_cur[IDX_TGAS]/1e5)) *exp(-4.73638e5/y_cur[IDX_TGAS])*y_cur[IDX_eM] - 5.54e-17 * pow(y_cur[IDX_TGAS], -.0397) / (1.0+sqrt(y_cur[IDX_TGAS]/1e5)) *exp(-4.73638e5/y_cur[IDX_TGAS])*y_cur[IDX_eM];
        data[jistart + 91] = 0.0 - 3.48e-26 * sqrt(y_cur[IDX_TGAS]) * pow(y_cur[IDX_TGAS]/1e3, -0.2) / (1.0+pow(y_cur[IDX_TGAS]/1e6, 0.7))*y_cur[IDX_eM];
        data[jistart + 92] = (gamma - 1.0) * (0.0 - 1.27e-21 * sqrt(y_cur[IDX_TGAS]) / (1.0 + sqrt(y_cur[IDX_TGAS]/1e5)) * exp(-1.578091e5/y_cur[IDX_TGAS])*y_cur[IDX_HI] - 9.38e-22 * sqrt(y_cur[IDX_TGAS]) / (1.0 + sqrt(y_cur[IDX_TGAS]/1e5)) * exp(-2.853354e5/y_cur[IDX_TGAS])*y_cur[IDX_HeI] - 4.95e-22 * sqrt(y_cur[IDX_TGAS]) / (1.0 + sqrt(y_cur[IDX_TGAS]/1e5)) * exp(-6.31515e5/y_cur[IDX_TGAS])*y_cur[IDX_HeII] - 5.01e-27 * pow(y_cur[IDX_TGAS], -0.1687) / (1.0 + sqrt(y_cur[IDX_TGAS]/1e5)) * exp(-5.5338e4/y_cur[IDX_TGAS])*y_cur[IDX_HeII]*y_cur[IDX_eM] - 5.01e-27 * pow(y_cur[IDX_TGAS], -0.1687) / (1.0 + sqrt(y_cur[IDX_TGAS]/1e5)) * exp(-5.5338e4/y_cur[IDX_TGAS])*y_cur[IDX_HeII]*y_cur[IDX_eM] - 8.7e-27 * sqrt(y_cur[IDX_TGAS]) * pow(y_cur[IDX_TGAS]/1e3, -0.2) / (1.0+pow(y_cur[IDX_TGAS]/1e6, 0.7))*y_cur[IDX_HII] - 1.24e-13 * pow(y_cur[IDX_TGAS], -1.5) * exp(-4.7e5/y_cur[IDX_TGAS]) * (1.0+0.3*exp(-9.4e4/y_cur[IDX_TGAS]))*y_cur[IDX_HeII] - 1.55e-26 * pow(y_cur[IDX_TGAS], 0.3647)*y_cur[IDX_HeII] - 3.48e-26 * sqrt(y_cur[IDX_TGAS]) * pow(y_cur[IDX_TGAS]/1e3, -0.2) / (1.0+pow(y_cur[IDX_TGAS]/1e6, 0.7))*y_cur[IDX_HeIII] - 9.1e-27 * pow(y_cur[IDX_TGAS], -0.1687) / (1.0+sqrt(y_cur[IDX_TGAS]/1e5)) * exp(-1.3179e4/y_cur[IDX_TGAS])*y_cur[IDX_HI]*y_cur[IDX_eM] - 9.1e-27 * pow(y_cur[IDX_TGAS], -0.1687) / (1.0+sqrt(y_cur[IDX_TGAS]/1e5)) * exp(-1.3179e4/y_cur[IDX_TGAS])*y_cur[IDX_HI]*y_cur[IDX_eM] - 5.54e-17 * pow(y_cur[IDX_TGAS], -.0397) / (1.0+sqrt(y_cur[IDX_TGAS]/1e5)) *exp(-4.73638e5/y_cur[IDX_TGAS])*y_cur[IDX_HeII] - 5.54e-17 * pow(y_cur[IDX_TGAS], -.0397) / (1.0+sqrt(y_cur[IDX_TGAS]/1e5)) *exp(-4.73638e5/y_cur[IDX_TGAS])*y_cur[IDX_HeII] ) / kerg / GetNumDens(y);
                // clang-format on
    }
}

/* */

int Fex(realtype t, N_Vector u, N_Vector udot, void *user_data) {
    /* */

    realtype *y         = N_VGetDeviceArrayPointer_Cuda(u);
    realtype *ydot      = N_VGetDeviceArrayPointer_Cuda(udot);
    NaunetData *h_udata = (NaunetData *)user_data;
    NaunetData *d_udata;

    // check the size of system (number of cells/ a batch)
    sunindextype lrw, liw;
    N_VSpace_Cuda(u, &lrw, &liw);
    int nsystem = lrw / NEQUATIONS;

    // copy the user data for each system/cell
    hipMalloc((void **)&d_udata, sizeof(NaunetData) * nsystem);
    hipMemcpy(d_udata, h_udata, sizeof(NaunetData) * nsystem,
               hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    unsigned block_size = min(BLOCKSIZE, nsystem);
    unsigned grid_size =
        max(1, min(MAXNGROUPS / BLOCKSIZE, nsystem / BLOCKSIZE));
    FexKernel<<<grid_size, block_size>>>(y, ydot, d_udata, nsystem);

    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess) {
        fprintf(stderr, ">>> ERROR in fex: hipGetLastError returned %s\n",
                hipGetErrorName(cuerr));
        return -1;
    }
    hipFree(d_udata);

    /* */

    return NAUNET_SUCCESS;
}

int Jac(realtype t, N_Vector u, N_Vector fu, SUNMatrix jmatrix, void *user_data,
        N_Vector tmp1, N_Vector tmp2, N_Vector tmp3) {
    /* */
    realtype *y         = N_VGetDeviceArrayPointer_Cuda(u);
    realtype *data      = SUNMatrix_cuSparse_Data(jmatrix);
    NaunetData *h_udata = (NaunetData *)user_data;
    NaunetData *d_udata;

    int nsystem = SUNMatrix_cuSparse_NumBlocks(jmatrix);

    hipMalloc((void **)&d_udata, sizeof(NaunetData) * nsystem);
    hipMemcpy(d_udata, h_udata, sizeof(NaunetData) * nsystem,
               hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    unsigned block_size = min(BLOCKSIZE, nsystem);
    unsigned grid_size =
        max(1, min(MAXNGROUPS / BLOCKSIZE, nsystem / BLOCKSIZE));
    JacKernel<<<grid_size, block_size>>>(y, data, d_udata, nsystem);

    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess) {
        fprintf(stderr, ">>> ERROR in jac: hipGetLastError returned %s\n",
                hipGetErrorName(cuerr));
        return -1;
    }
    hipFree(d_udata);

    /* */

    return NAUNET_SUCCESS;
}